#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2016, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#include "lbann/layers/loss/top_k_categorical_accuracy.hpp"
#include "lbann/utils/cuda.hpp"
#include "lbann/utils/exception.hpp"

#include <thrust/system/cuda/execution_policy.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>

namespace lbann {

namespace {

/** Sparse vector entry. */
struct entry {

  /** Vector entry value. */
  DataType value;
  /** Vector entry index. */
  El::Int index;

  /** Minimum possible value. */
  static constexpr DataType min_value = -std::numeric_limits<DataType>::infinity();
  /** Maximum possible index. */
  static constexpr El::Int max_index = std::numeric_limits<El::Int>::max();

};

/** Comparison operation to sort sparse vector entries.
 *  Entries are sorted by value in decreasing order. Entries with the
 *  same value are sorted by index in increasing order.
 */
struct entry_compare : thrust::binary_function<entry,entry,bool> {
  __host__ __device__ bool operator()(const entry& a, const entry& b) const {
    return a.value > b.value || (a.value == b.value && a.index < b.index);
  }
};

/** Initialize sparse vector entries from local input data.
 *  Each column of the input matrix is converted into a sparse
 *  vector.
 */
__global__ void initialize_local_entries(El::Int num_local_entries,
                                         El::Int local_height,
                                         El::Int local_width,
                                         El::Int col_shift,
                                         El::Int col_stride,
                                         const DataType* __restrict__ local_input,
                                         El::Int local_input_ldim,
                                         entry*  __restrict__ local_entries,
                                         El::Int* __restrict__ local_entry_cols) {
  const El::Int gid = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int num_threads = blockDim.x * gridDim.x;
  const El::Int num_local_entries_per_col = num_local_entries / local_width;
  for (El::Int i = gid; i < num_local_entries; i += num_threads) {
    const auto& row = i % num_local_entries_per_col;
    const auto& col = i / num_local_entries_per_col;
    if (row < local_height) {
      const auto& global_row = col_shift + row * col_stride;
      local_entries[i].value = local_input[row + col * local_input_ldim];
      local_entries[i].index = global_row;
    } else {
      local_entries[i].value = entry::min_value;
      local_entries[i].index = entry::max_index;
    }
    local_entry_cols[i] = col;
  }  
}

/** Fill an array with tensor dimension indices.
 *  Entries in 'indices' are populated with the dimension index for a
 *  corresponding entry in a packed tensor.
 */
__global__ void fill_tensor_indices(El::Int tensor_size,
                                    El::Int dim_max,
                                    El::Int dim_stride,
                                    El::Int* indices) {
  const El::Int gid = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int num_threads = blockDim.x * gridDim.x;
  for (El::Int i = gid; i < tensor_size; i += num_threads) {
    indices[i] = (i / dim_stride) % dim_max;
  }  
}

/** Set selected entries in local output data to one. */
__global__ void indicate_local_entries(El::Int k,
                                       El::Int num_entries,
                                       El::Int height,
                                       El::Int local_height,
                                       El::Int local_width,
                                       El::Int col_rank,
                                       El::Int col_align,
                                       El::Int col_shift,
                                       El::Int col_stride,
                                       DataType* __restrict__ local_output,
                                       El::Int local_output_ldim,
                                       const entry*  __restrict__ entries,
                                       El::Int entries_ldim) {
  const El::Int gid = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int num_threads = blockDim.x * gridDim.x;
  for (El::Int i = gid; i < num_entries; i += num_threads) {
    const auto& ind = i % k;
    const auto& col = i / k;
    const auto& global_row = entries[ind + col * entries_ldim].index;
    const auto& row_owner = (global_row + col_align) % col_stride;
    if (global_row < height && row_owner == col_rank) {
      const auto& row = (global_row > col_shift ?
                         (global_row - col_shift - 1) / col_stride + 1 :
                         0);
      local_output[row + col * local_output_ldim] = DataType(1);
    }
  }  
}

/** GPU implementation of top-k categorical accuracy layer forward prop. */
void fp_gpu(lbann_comm& comm,
            El::Int k, const AbsDistMat& input, AbsDistMat& output) {
  if (input.Wrap() != El::ELEMENT || output.Wrap() != El::ELEMENT) {
    LBANN_ERROR("top-k categorical accuracy layer GPU implementation assumes elemental "
                "distributed matrices");
  }

  // Local matrices
  const auto& local_input = input.LockedMatrix();
  auto& local_output = output.Matrix();
  const El::Int height = input.Height();
  const El::Int local_height = local_input.Height();
  const El::Int local_width = local_input.Width();

  // Trivial cases
  if (k < 1) {
    El::Zero(output);
    return;
  } else if (k >= height) {
    El::Fill(output, DataType(1));
    return;
  } else if (local_width < 1) {
    return;
  }
  
  // Column communicator
  auto&& col_comm = input.ColComm();
  const auto& col_comm_rank = El::mpi::Rank(col_comm);
  const auto& col_comm_size = El::mpi::Size(col_comm);

  // GPU objects
  using entry_array = El::Memory<entry, El::Device::GPU>;
  using entry_ptr = thrust::device_ptr<entry>;
  using index_array = El::Memory<El::Int, El::Device::GPU>;
  using index_ptr = thrust::device_ptr<El::Int>;
#ifdef HYDROGEN_HAVE_CUB
  const unsigned int memory_mode = 1; // CUB GPU memory pool
#else
  const unsigned int memory_mode = 0;
#endif // HYDROGEN_HAVE_CUB
  auto&& stream = El::GPUManager::Stream();
  entry_array top_entries(col_comm_size * local_width * k, memory_mode);
  auto* local_top_entries = (top_entries.Buffer()
                             + col_comm_rank * local_width * k);

  // Find top-k entries in each local matrix column
  {
    const auto& num_local_entries_per_col = std::max(local_height, k);
    const auto& num_local_entries = local_width * num_local_entries_per_col;
    const auto& block_dim = 256;
    const auto& grid_dim = (num_local_entries + block_dim - 1) / block_dim;
    entry_array local_entries(num_local_entries, memory_mode);
    index_array local_entry_cols(num_local_entries, memory_mode);
    initialize_local_entries<<<grid_dim, block_dim, 0, stream>>>(
      num_local_entries, local_height, local_width,
      input.ColShift(), input.ColStride(),
      local_input.LockedBuffer(), local_input.LDim(),
      local_entries.Buffer(), local_entry_cols.Buffer());
    thrust::sort_by_key(thrust::cuda::par.on(stream),
                        entry_ptr(local_entries.Buffer()),
                        entry_ptr(local_entries.Buffer() + num_local_entries),
                        index_ptr(local_entry_cols.Buffer()),
                        entry_compare());
    thrust::stable_sort_by_key(thrust::cuda::par.on(stream),
                               index_ptr(local_entry_cols.Buffer()),
                               index_ptr(local_entry_cols.Buffer() + num_local_entries),
                               entry_ptr(local_entries.Buffer()));
    CHECK_CUDA(hipMemcpy2DAsync(local_top_entries,
                                 k * sizeof(entry),
                                 local_entries.Buffer(),
                                 num_local_entries_per_col * sizeof(entry),
                                 k * sizeof(entry),
                                 local_width,
                                 hipMemcpyDeviceToDevice,
                                 stream));
  }

  // Find top-k entries in each global matrix column
  if (col_comm_size > 1) {
    const auto& num_entries_per_rank = local_width * k;
    const auto& num_entries = col_comm_size * num_entries_per_rank;
    const auto& block_dim = 256;
    const auto& grid_dim = (num_entries + block_dim - 1) / block_dim;
    comm.all_gather(reinterpret_cast<El::byte*>(MPI_IN_PLACE),
                    num_entries_per_rank * sizeof(entry),
                    reinterpret_cast<El::byte*>(top_entries.Buffer()),
                    num_entries_per_rank * sizeof(entry),
                    col_comm);
    index_array top_entry_cols(num_entries, memory_mode);
    fill_tensor_indices<<<grid_dim, block_dim, 0, stream>>>(
      num_entries, local_width, k, top_entry_cols.Buffer());
    thrust::sort_by_key(thrust::cuda::par.on(stream),
                        entry_ptr(top_entries.Buffer()),
                        entry_ptr(top_entries.Buffer() + num_entries),
                        index_ptr(top_entry_cols.Buffer()),
                        entry_compare());
    thrust::stable_sort_by_key(thrust::cuda::par.on(stream),
                               index_ptr(top_entry_cols.Buffer()),
                               index_ptr(top_entry_cols.Buffer() + num_entries),
                               entry_ptr(top_entries.Buffer()));
  }

  // Indicate output entries corresponding to top-k input entries
  El::Zero(output);
  if (output.Participating() && local_height > 0 && local_width > 0) {
    const auto& num_entries = local_width * k;
    const auto& block_dim = 256;
    const auto& grid_dim = (num_entries + block_dim - 1) / block_dim;
    indicate_local_entries<<<grid_dim, block_dim, 0, stream>>>(
      k, num_entries, height, local_height, local_width,
      output.ColRank(), output.ColAlign(),
      output.ColShift(), output.ColStride(),
      local_output.Buffer(), local_output.LDim(),
      top_entries.Buffer(), col_comm_size * k);
  }

}

} // namespace

template <>
void top_k_categorical_accuracy_layer<data_layout::MODEL_PARALLEL, El::Device::GPU>
     ::fp_compute() {
  LBANN_ERROR("not yet implemented");
}
template <>
void top_k_categorical_accuracy_layer<data_layout::DATA_PARALLEL, El::Device::GPU>
     ::fp_compute() {
  LBANN_ERROR("not yet implemented");
}

} // namespace lbann
