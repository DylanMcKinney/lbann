#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2016, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
//
// cudnn_wrapper .hpp .cpp - cuDNN support - wrapper classes, utility functions
////////////////////////////////////////////////////////////////////////////////

#include "lbann/utils/cudnn_wrapper.hpp"
#include "lbann/utils/exception.hpp"

#include "El.hpp"

using namespace cudnn;
using namespace lbann;

namespace cudnn {

namespace {

__global__ void reduce_kernel(DataType *dst, const DataType *src,
                              El::Int len) {
  int offset = blockIdx.x * blockDim.x + threadIdx.x;
  if (offset >= len) return;
  dst[offset] += src[offset];
}

__global__ void scale_kernel(DataType *data, 
                             const DataType scale,
                             El::Int len) {

  int offset = blockIdx.x * blockDim.x + threadIdx.x;
  if (offset >= len) return;
  data[offset] *= scale;

}

}

void cudnn_manager::allreduce_on_gpus(std::vector<DataType*>& gpu_data,
                                      El::Int height,
                                      El::Int width) {
  if (m_num_gpus < 2) {
    return;
  }

  if(is_nccl_used()){
#ifndef __LIB_NCCL
    throw lbann::lbann_exception("cudnn_wrapper_cuda: NCCL is not used");
#else
    /// Use NCCL implementation if available
    allreduce_on_gpus_nccl(gpu_data, height, width);
    return;
#endif // __LIB_NCCL
  } else{


    const El::Int buf_len = 1 << 27;
    const El::Int work_len = buf_len * 2; // double buffering
    const El::Int work_len_bytes = work_len * sizeof(DataType);

    std::vector<DataType*> bufs[2];
    for(int i=0; i<m_num_gpus; ++i) {
      if (get_work_space_size(i) < work_len_bytes) {
        set_work_space_size(i, work_len_bytes); 
      }
      bufs[0].push_back(static_cast<DataType*>(get_work_space(i)));
      bufs[1].push_back(static_cast<DataType*>(get_work_space(i)) + buf_len);
    }  


    El::Int total_len = height * width;
    El::Int offset = 0;

    do {
      El::Int len = std::min(total_len - offset, buf_len);
      int sbuf_idx = 0;
      int dbuf_idx = 1;
      for (int j = 0; j < m_num_gpus - 1; ++j) {
        for(int i = 0; i < m_num_gpus; ++i) {
          CHECK_CUDA(hipSetDevice(m_gpus[i]));
          int src_dev = i;
          int dst_dev = (i + 1) % m_num_gpus;              
          DataType *src_buf = j == 0 ? gpu_data[src_dev] + offset : bufs[sbuf_idx][src_dev];
          DataType *dst_buf = bufs[dbuf_idx][dst_dev];
          // copy to the next device in the ring
          FORCE_CHECK_CUDA(hipMemcpyPeerAsync(dst_buf, dst_dev, src_buf, src_dev,
                                             len * sizeof(DataType), get_stream(src_dev)));
        }
        synchronize();
        for(int i = 0; i < m_num_gpus; ++i) {
          CHECK_CUDA(hipSetDevice(m_gpus[i]));        
          DataType *dst_buf = bufs[dbuf_idx][i];
          // TODO: use Thrust
          int tb_dim = 256;
          int grid_dim = len / tb_dim + (len % tb_dim ? 1 : 0);
          reduce_kernel<<<grid_dim, tb_dim>>>(gpu_data[i] + offset, dst_buf, len);
        }
        std::swap(sbuf_idx, dbuf_idx);
      }
      offset += len;
    } while (offset < total_len);
  }
}

/// @todo Efficient implementation
void cudnn_manager::global_allreduce_on_gpus(std::vector<DataType*>& gpu_data,
                                             El::Int height,
                                             El::Int width,
                                             El::mpi::Comm comm) {
  static Mat cpu_workspace;
  cpu_workspace.Resize(height, width);
  allreduce_on_gpus(gpu_data, height, width);
  copy_from_gpu(0, cpu_workspace, gpu_data[0]);
  synchronize();
  El::AllReduce(cpu_workspace, comm);
  broadcast_to_gpus(gpu_data, cpu_workspace);
}

#ifdef __LIB_NCCL
/// Convert DataType to NCCL data type. DataType is either double or float (default).
ncclDataType_t cudnn_manager::nccl_datatype() {
  switch(sizeof(DataType) ) {
    case 8:
      return ncclDouble;
    case 4:
      return ncclFloat;
    case 2:
      return ncclHalf;
    default:
      throw lbann::lbann_exception("cudnn_wrapper_cuda: invalid data type for NCCL");
  }
}

void cudnn_manager::allreduce_on_gpus_nccl(std::vector<DataType*>& gpu_data,
                                           El::Int height,
                                           El::Int width,
                                           DataType scale) {
/**
  gpu_data is a vector of pointers, each of which points to a part of
  matrix allocated to GPU memory. Since we assume that one MPI rank is
  assigned to one GPU, the number of element in gpu_data is 1. */

  /// It is assumed each MPI rank is assigned to one GPU (that is, m_num_gpus==1)

  if(m_num_gpus != 1){
    char line[1024];
    sprintf(line, "cudnn_wrapper_cuda: the number of GPUs assigned to process is %d; should be 1", m_num_gpus);
    throw lbann::lbann_exception(line);
    //throw lbann::lbann_exception("cudnn_wrapper_cuda: the number of GPUs assigned to process is not 1");

  }

  int local_rank = comm->get_rank_in_node();
  ncclDataType_t type = nccl_datatype();
  El::Int total_len = height * width;

  //DataType *target_buffer;
  //CHECK_CUDA(hipSetDevice(local_rank));

  for(int i = 0; i < m_num_gpus; ++i) {

    CHECK_CUDA(hipSetDevice(m_gpus[i]));
    //CHECK_CUDA(hipMalloc((void **) &target_buffer, total_len*sizeof(DataType)));
    //NCCLCHECK(ncclAllReduce(gpu_data[0], target_buffer, total_len, type, ncclSum, m_nccl_comm, get_stream(local_rank)));
    NCCLCHECK(ncclAllReduce(gpu_data[0], gpu_data[0], total_len, type, ncclSum, m_nccl_comm, get_stream(local_rank)));
    //CHECK_CUDA(hipMemcpy(gpu_data[0], target_buffer, total_len*sizeof(type),  hipMemcpyDeviceToDevice));
    //CHECK_CUDA(hipFree ((void **) &target_buffer));

    /// Apply scaling, if scale != 1
    if(scale != DataType(1)) {
      int tb_dim = 256;
      int grid_dim = total_len/tb_dim + (total_len % tb_dim ? 1 : 0);
      scale_kernel<<<grid_dim, tb_dim>>>(gpu_data[0], scale, total_len);
    }
  }

}
#endif // __LIB_NCCL

} // namespace cudnn
